#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <chrono> // Used for time measurements
#include <cstring>
#include <hip/hip_runtime.h>

std::vector<char> read_file(const char* filename)
{
    // Open the file in binary mode
    std::ifstream file(filename, std::ios::binary);

    // Check if the file opened successfully
    if (!file) {
        std::cerr << "Error: Could not open the file " << filename << std::endl;
        return {};
    }

    // Move the file cursor to the end of the file to get its size
    file.seekg(0, std::ios::end);
    std::streamsize fileSize = file.tellg();

    // Return the file cursor to the beginning of the file
    file.seekg(0, std::ios::beg);

    // Create a vector of the same size as the file to hold the content
    std::vector<char> buffer(fileSize);

    // Read the entire file into the vector
    if (!file.read(buffer.data(), fileSize)) {
        std::cerr << "Error: Could not read the file content." << std::endl;
        return {};
    }

    // Close the file
    file.close();

    // Output the number of bytes read
    std::cout << "Successfully read " << buffer.size() << " bytes from the file." << std::endl;

    // Convert to lowercase
    std::transform(buffer.begin(), buffer.end(), buffer.begin(), [](char c) { return std::tolower(c); });

    return buffer;
}

// Custom string comparison function for CUDA (works like strncmp)
__device__ bool compare_token(const char* data, const char* token, int token_len)
{
    for (int i = 0; i < token_len; ++i)
    {
        if (data[i] != token[i])
        {
            return false; // If any character doesn't match, return false
        }
    }
    return true;
}

// CUDA kernel to calculate token occurrences
__global__ void calc_token_occurrences_cuda(const char* data, int data_size, const char* token, int token_len, int* result)
{
    int numOccurrences = 0;

    // Single thread for simplicity
    for (int i = 0; i < data_size; ++i)
    {
        // test 1: does this match the token?
        if (!compare_token(&data[i], token, token_len))
            continue;

        // test 2: is the prefix a non-letter character?
        auto iPrefix = i - 1;
        if (iPrefix >= 0 && data[iPrefix] >= 'a' && data[iPrefix] <= 'z')
            continue;

        // test 3: is the suffix a non-letter character?
        auto iSuffix = i + token_len;
        if (iSuffix < data_size && data[iSuffix] >= 'a' && data[iSuffix] <= 'z')
            continue;

        // Increment occurrence count
        numOccurrences++;
    }

    // Write result back to host
    *result = numOccurrences;
}

int main()
{
    // Example chosen file
    const char* filepath = "dataset/shakespeare.txt";
    std::ofstream data("dataUnop.csv", std::ofstream::out);
    double totalDuration = 0.0;
     
    std::vector<char> file_data = read_file(filepath);
    if (file_data.empty())
        return -1;
    // Example word list
    const char* words[] = { "sword", "fire", "death", "love", "hate", "the", "man", "woman" };
    
    //get the file data size for device allocation
    int data_size = file_data.size();

    // Start total timer
    auto totalStart = std::chrono::high_resolution_clock::now();

    // Allocate device memory for the file data
    char* d_data;
    hipMalloc((void**)&d_data, data_size * sizeof(char));
    //copy the file data from host memory to device memory 
    hipMemcpy(d_data, file_data.data(), data_size * sizeof(char), hipMemcpyHostToDevice);

    

    for (const char* word : words)
    {
        int token_len = strlen(word);
        int* d_occurrences;
        int occurrences;

        // Allocate memory for the result on the device
        hipMalloc((void**)&d_occurrences, sizeof(int));

        // Copy the token to the device
        char* d_token;
        hipMalloc((void**)&d_token, token_len * sizeof(char));
        hipMemcpy(d_token, word, token_len * sizeof(char), hipMemcpyHostToDevice);

        //start timer ---------------------------------------------------------------------------------
        auto start = std::chrono::high_resolution_clock::now();
        // Launch CUDA kernel with 1 block and 1 thread (single-threaded execution)
        calc_token_occurrences_cuda << <1, 1 >> > (d_data, data_size, d_token, token_len, d_occurrences);
        //nd timer ---------------------------------------------------------------------------------
        auto end = std::chrono::high_resolution_clock::now();

        // Copy the result back to the host
        hipMemcpy(&occurrences, d_occurrences, sizeof(int), hipMemcpyDeviceToHost);

       
        // Free the device memory for this word
        hipFree(d_token);
        hipFree(d_occurrences);

        std::cout << "Found " << occurrences << " occurrences of word: " << word << std::endl;
        
        std::chrono::duration<double> duration = end - start;
        totalDuration += duration.count();
        data << duration.count() << std::endl;
    }

        double averageDuration = totalDuration; // / numRuns;

    // Print average duration
    std::cout << "Average CUDA execution time (" << 1 << " times run): " << averageDuration << " seconds" << std::endl;

    // Free device memory
    hipFree(d_data);

    data.close();

    return 0;
}
